#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include ""
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "time.h"
#include <hip/hip_vector_types.h>
//#include <iostream>
//#include <string>

unsigned int K, S, H, L;
unsigned int **GLDPC;
char ss[17];
char **s;
unsigned int *snumone;
unsigned int countdfs = 0;

void dfs(unsigned int i)
{
	unsigned int j;
	//int x = pow(2., (int)H);
    if(i == H) 
    {
    	for(j = 0; j < H; j++)
    	{
    		s[countdfs][j] = ss[j];
    		if(48 == s[countdfs][j])s[countdfs][j] = 0;
    		if(49 == s[countdfs][j])s[countdfs][j] = 1;
    	}
    	//printf("%s\n", ss); //����ײ������
    	//s[i] = ss;
    	countdfs++;
    }
    else
    {
        dfs(i + 1); //�����������
        if(ss[i] == '0') ss[i] = '1';
        else ss[i] = '0'; //�ѵ�ǰλ��ת
        dfs(i + 1); //������������
    }
}


void gen_graycode()
{
	/*
	int i, j;
	int x = pow(2., n);
	//string* s=new string[x];
	char **s;
	s = (char**)malloc(sizeof(char*) * x);
	for(i = 0; i < x; i++)
	{
		s[i] = (char*)malloc(sizeof(char) * n);
	}
	unsigned int *snumone;
	snumone = (unsigned int*)malloc(sizeof(unsigned int) * x);
       for(i = 0; i < x; i++)
    	{
    		for(j = 0; j < n; j++)
    		{
    			s[i][j] = 0;
    		}
    	}
    	for(i = 0; i < x; i++)
    	{
    		snumone[i] = 0;
    	}

	int a = 0, b = 1;
    	for(i = 1; i < x; i++)
    	{
    		for(j = 0; j < n; j++)s[i][j] = s[i - 1][j];
    		if(0 != (i % 2))
    		{
    			printf("b = 0\n");
    			s[i][n - 1] = (s[i - 1][n - 1] + 1) % 2;
    		}
    		else
    		{
    		if(1 == b)
    		{
    			a++;
    			b = a;
    			printf("b = %d\n", b);
    			s[i][n - 1 - b] = (s[i - 1][n - 1 - b] + 1) % 2;
    		}
    		else
    		{
    			b--;
    			printf("b = %d\n", b);
    			s[i][n - 1 - b] = (s[i - 1][n - 1 - b] + 1) % 2;
    		}
    	}
    	for(j = 0; j < n; j++)
		{
			printf("%d ", s[i][j]);
		}
		printf("\n");
    	}

	for(i = 0; i < x; i++)
	{
		for(j = 0; j < n; j++)
		{
			if(1 == s[i][j])snumone[i]++;
		}
	}

	printf("-----------------------------\n");
	unsigned int halfH = H / 2;
	for(i = 0; i < x; i++)
	{
		if(halfH == snumone[i])
		{
			for(j = 0; j < n; j++)
			{
				printf("%d ", s[i][j]);
			}
			printf("\n");
		}
	}

    	free(s);
    	s = NULL;
    	free(snumone);
    	snumone = NULL;
       */

	unsigned int i, j;
	
	unsigned int x = (unsigned int)pow(2., (int)H);
	
	s = (char**)malloc(sizeof(char*) * x);
	for(i = 0; i < x; i++)
	{
		s[i] = (char*)malloc(sizeof(char) * H);
	}
	
	snumone = (unsigned int*)malloc(sizeof(unsigned int) * x);
    for(i = 0; i < x; i++)
    {
   		for(j = 0; j < H; j++)
   		{
   			s[i][j] = 0;
   		}
    }
    for(i = 0; i < x; i++)
    {
   		snumone[i] = 0;
   	}
	//scanf("%d",&n);
    //if(!H) break;
    for(i = 0; i < 17; i++) ss[i] = '0';
    ss[H] = '\0';
    dfs(0);
    //printf("\n");

    //printf("-----------------------------\n");
    for(i = 0; i < x; i++)
	{
		for(j = 0; j < H; j++)
		{
			if(1 == s[i][j])snumone[i]++;
			//printf("%d", s[i][j]);
		}
		//printf("\n");
	}
	
	unsigned int halfH = H / 2;
	//printf("%d\n", halfH);
	for(i = 0; i < x; i++)
	{
		//printf("%d\n", halfH);
		if(halfH == snumone[i])
		{
			for(j = 0; j < H; j++)
			{
				printf("%d ", s[i][j]);
			}
			printf("\n");
		}
	}
	
}


void GParameters()
{
	unsigned int i;
	long long Ha, Hb, Hc, Hd, He, Hf;
	
	printf("Please Input the K: ");
	scanf("%d", &K);

	unsigned int F = 1;
	while(1)
	{
		if(F * (F - 1) > (2 * K))break;
		F++;
	}
	S = (unsigned int)ceil(0.01 * K) + F;
	printf("S0: %d\n", S);
	while(1)
	{
		for(i = 2; i < (S - 1); i++)
		{
			if(0 == (S % i))i = S + 1;
		}
		printf("i: %d\n", i);
		if((S - 1) == i)break;
		S++;
		printf("S: %d\n", S);
	}

	H = 2;
	while(1)
	{
		Ha = (unsigned int)ceil(float(H / 2));
		Hb = H - Ha;
		Hc = 1;
		Hd = 1;
		He = 1;
		Hf = 0;
		
		for(i = 1; i <= H;i++)
		{
			Hc = i * Hc;
		}
		for(i = 1; i <= Ha;i++)
		{
			Hd = i * Hd;
		}
		for(i = 1; i <= Hb;i++)
		{
			He = i * He;
		}
		/*
		printf("Hc: %d\n", Hc);
		printf("Hd: %d\n", Hd);
		printf("He: %d\n", He);
		*/

		Hf = Hc / (Hd * He);
		if(Hf > (K + S))break;
		H++;
	}

	L = K + S + H;
}

void GMatrix()
{
	unsigned int i, j, k;

	unsigned int mod = K % S;
	unsigned int numblockldpc = (K - mod) / S + 1;
	printf("numblockldpc: %d\n", numblockldpc);
	unsigned int Indexa, Indexb, Indexc;
	for(k = 0; k < (numblockldpc - 1); k++)
	{
		Indexa = 0;
		Indexb = k + 1;
		Indexc = 2 * k + 2;
		GLDPC[Indexa][k * S] = 1;
		GLDPC[Indexb][k * S] = 1;
		GLDPC[Indexc][k * S] = 1;
		/*
		printf("!!!\n");
		printf("a: %d\n", Indexa);
		printf("b: %d\n", Indexb);
		printf("c: %d\n", Indexc);
		*/
		for(j = 1; j < S; j++)
		{
			Indexa = (Indexa + 1) % S;
			Indexb = (Indexb + 1) % S;
			Indexc = (Indexc + 1) % S;
			/*
			printf("a: %d\n", Indexa);
			printf("b: %d\n", Indexb);
			printf("c: %d\n", Indexc);
			*/
			GLDPC[Indexa][k * S + j] = 1;
			GLDPC[Indexb][k * S + j] = 1;
			GLDPC[Indexc][k * S + j] = 1;
		}
	}
	k = (numblockldpc - 1);
	Indexa = 0;
	Indexb = k + 1;
	Indexc = 2 * k + 2;
	GLDPC[Indexa][k * S] = 1;
	GLDPC[Indexb][k * S] = 1;
	GLDPC[Indexc][k * S] = 1;
	for(j = 1; j < mod; j++)
	{
		Indexa = (Indexa + 1) % S;
		Indexb = (Indexb + 1) % S;
		Indexc = (Indexc + 1) % S;
		GLDPC[Indexa][k * S + j] = 1;
		GLDPC[Indexb][k * S + j] = 1;
		GLDPC[Indexc][k * S + j] = 1;
	}
	for(i = 0; i < S; i++)
	{
		for(j = 0; j < K; j++)
		{
			printf("%d ", GLDPC[i][j]);
		}
		printf("\n");
	}
}

void init()
{
	unsigned int i, j;
	
	GLDPC = (unsigned int**)malloc(sizeof(unsigned int*) * S);
	for (i = 0; i < S; i++)
	{
  		GLDPC[i] = (unsigned int*)malloc(sizeof(unsigned int) * K);
  	}
  	for(i = 0; i < S; i++)
	{
		for(j = 0; j < K; j++)
		{
			GLDPC[i][j] = 0;
		}
	}
}

void Exit()
{
	unsigned int i;
	unsigned int x = (unsigned int)pow(2., (int)H);
	
	for (i = 0; i < S; i++)
	{
  		free(GLDPC[i]);
		GLDPC[i] = NULL;
  	}
	free(GLDPC);
	GLDPC = NULL;
	for(i = 0; i < x; i++)
	{
		free(s[i]);
		s[i] = NULL;
	}
	free(s);
    s = NULL;
    free(snumone);
    snumone = NULL;
}

int main()
{
	GParameters();
	printf("S: %d\n", S);
	printf("H: %d\n", H);
	printf("L: %d\n", L);
	init();
	GMatrix();
	gen_graycode();

	Exit();
}
